﻿
#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <stdio.h>
using namespace std;
#include <string>
#include <vector>




__global__ void bajarBloquesKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int* generar_matriz(int filas, int columnas, int maximo) {

    int* tablero = new int[filas * columnas];

    srand(time(NULL));

    // Generar números aleatorios y almacenarlos en la matriz
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            tablero[i * columnas + j] = rand() % maximo + 1;
        }
    }

    return tablero;
}

// definición de la función
string seleccionarConfiguracion() {
    char ejecucion;
    int nivel, filas, columnas;
    string salida;
    cout << "Introduce el modo de ejecucion: ";
    cin >> ejecucion;
    cout << "Introduce el nivel de dificultad (1 o 2): ";
    cin >> nivel;
    cout << "Introduce el numero de filas del tablero: ";
    cin >> filas;
    cout << "Introduce el nivel numero de columnas del tablero: ";
    cin >> columnas;
    salida = string(1, ejecucion) + to_string(nivel) + to_string(filas) + to_string(columnas);
    return salida;
}

void print_matriz(int* matriz, int filas, int columnas) {
    for (int i = 0; i < (filas + 1); i++) {
        for (int j = 0; j < (columnas + 1); j++) {
            if (j == 0) {
                printf("%d ", i);

            }
            else if (i == 0) {
                printf("%d ", j);

            }
            else {
                printf("%d ", matriz[((i - 1) * columnas) + (j - 1)]);
            }
            //printf("%d ", matriz[i * columnas + j]);
        }
        printf("\n");
    }
    printf("\n");
}




int main() {

    int vidas, maximo, fila, columna = 5;

    // llamada a la función y guardado del resultado en una variable
    string salida = seleccionarConfiguracion();

    // impresión de la salida por pantalla
    cout << "La configuración elegida es: " << salida << endl;

    if (salida[1] == '1') {
        maximo = 4;
    }
    else {
        maximo = 6;
    }
    //Se resta 48 al valor de salida[x] para cambiar el valor de ascii al número que nos interesa
    int filas = salida[2] - 48;
    int columnas = salida[3] - 48;

    // Generar la matriz y mostrarla por pantalla
    int* tablero = generar_matriz(filas, columnas, maximo);

    print_matriz(tablero, filas, columnas);


    /*while (vidas > 0) {
        print_matriz(tablero, filas, columnas);
        printf("Haga la fila seleccionada")
        

    }*/

    return 0;
}



